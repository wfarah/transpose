
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdint.h>

#include <sys/mman.h>

#define ELAPSED_NS(start,stop) \
  (((int64_t)stop.tv_sec-start.tv_sec)*1000*1000*1000+(stop.tv_nsec-start.tv_nsec))

#define USE_MUTLI_THREAD

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct
{
  size_t ntime;
  size_t obsnchan;
  size_t nbits;
  size_t ndim;
  size_t npol;
  size_t piperblk;

  size_t itime_packets;
  size_t istride;
  size_t tstride;
  size_t ostride;
} db_transpose_t;


__global__
void transpose(db_transpose_t * ctx, const void* in, void* out)
{
  // To be used in pointer arithmetic
  const char* inbuf;
  char* outbuf;

  const char* baseinbuf = (const char*) in;
  char* baseoutbuf      = (char*) out;

  //size_t itime_packets = ctx->itime_packets;
  size_t istride = ctx->istride;
  size_t tstride = ctx->tstride;
  size_t ostride = ctx->ostride;

  size_t ichan;
  size_t iptime = blockIdx.x;
  size_t nchans = ctx->obsnchan;

  size_t nloops = (ctx->obsnchan + blockDim.x - 1)/blockDim.x;

  for (int igrid=0; igrid < nloops; igrid++)
  {
      ichan = threadIdx.x + blockDim.x * igrid;

      if (ichan < nchans)
      {
          inbuf  = baseinbuf  + iptime*tstride + ichan*istride;
          outbuf = baseoutbuf + iptime*istride + ichan*ostride;
          memcpy(outbuf, inbuf, istride);
      }
  }

  /*
  // Loop over entire spectrum-packets over all the chans
//#pragma omp parallel for private (inbuf, outbuf)
  for (size_t iptime=0; iptime < itime_packets; iptime++)
  {
    inbuf  = baseinbuf + iptime*tstride;
    outbuf = baseoutbuf + iptime*istride;
    for (size_t ichan=0; ichan < ctx->obsnchan; ichan++)
    {
      memcpy(outbuf, inbuf, istride);
      inbuf += istride;
      outbuf += ostride;
    }
  }
  */
}





int test_buf(char* outbuf, size_t nsamps)
{
  for (size_t i=0; i< nsamps; i++)
  {
    if ( outbuf[i] != 10 )
    {
      fprintf(stderr, "Test failed at sample: %li, value: %i\n", i, outbuf[i]);
      return EXIT_FAILURE;
    }
  }
  return 0;
}


int main(int argc, char* argv[])
{
  size_t NTIME    = 16; //ntime samples per packet
  size_t OBSNCHAN = 2048; //total number of channels (all the antennas)
  size_t NBITS    = 4; //Number of bits
  size_t NDIM     = 2; //i.e. complex
  size_t NPOL     = 2; //Number of polarisations
  size_t PIPERBLK = 8192*8; // Number of time samples in a block

  //size_t NTHREADS = 1;
  //omp_set_num_threads(NTHREADS);

  // create context
  db_transpose_t* ctx;
  hipMallocManaged((void**)&ctx, sizeof(*ctx) );

  ctx->ntime = NTIME;
  ctx->obsnchan = OBSNCHAN;
  ctx->nbits = NBITS;
  ctx->ndim = NDIM;
  ctx->npol = NPOL;
  ctx->piperblk = PIPERBLK;

  // number of packets that span the entire data block, in time
  ctx->itime_packets = ctx->piperblk / ctx->ntime;

  // bytes to stride within a packet in input buffer
  // and also amount to copy at a time
  ctx->istride = (ctx->npol * ctx->ndim * ctx->nbits * ctx->ntime)/8;

  ctx->tstride = ctx->obsnchan * ctx->istride;
  ctx->ostride = ctx->itime_packets * ctx->istride;

  fprintf(stderr, "itime packets: %li, obsnchan: %li\n",
          ctx->itime_packets, ctx->obsnchan);


  //Create input and output buffers, same size
  size_t buffsize = (NPOL * NDIM * NBITS * PIPERBLK * OBSNCHAN)/8; // Bytes

  fprintf(stderr, "BUFFSIZE: %li\n", buffsize); 

  char* outbuf = (char*) malloc(buffsize);
  char* inbuf = (char*) malloc(buffsize);

  for (size_t i=0; i<buffsize; i++)
  {
      outbuf[i] = 0;
      inbuf[i]  = 10;
  }

  // GPU memory
  char *d_inbuf, *d_outbuf;
  if (hipMalloc((void**) &d_inbuf, buffsize) != hipSuccess)
  {
      fprintf(stderr, "Couldn't allocate inbuf\n");
      return -1;
  }
  if (hipMalloc((void**) &d_outbuf, buffsize) != hipSuccess)
  {
      fprintf(stderr, "Couldn't allocate outbuf\n");
      return -1;
  }

  struct timespec start, stop;

  // Time the function
  clock_gettime(CLOCK_MONOTONIC, &start);
  int nloops = 60;
  for (size_t i = 0; i<nloops; i++)
  {
      if (hipMemcpy(d_inbuf, inbuf, buffsize, hipMemcpyHostToDevice) != hipSuccess)
      {
          fprintf(stderr, "Couldn't move data to device\n");
          return -1;
      }
      //transpose <<<ctx->itime_packets, 512>>> (ctx, d_inbuf, d_outbuf);
      transpose <<<ctx->itime_packets, 1024>>> (ctx, d_inbuf, d_outbuf);
      //cudaMemcpy(d_outbuf, d_inbuf, buffsize, cudaMemcpyDeviceToDevice);
      hipDeviceSynchronize();

      hipError_t error = hipGetLastError();
      if (error != hipSuccess)
      {
          fprintf(stderr, "Couldn't move data back to host, retval: %s\n", hipGetErrorString(error));
          return -1;
      }
      hipMemcpy(outbuf, d_outbuf, buffsize, hipMemcpyDeviceToHost); 
  }

  clock_gettime(CLOCK_MONOTONIC, &stop);

  uint64_t elapsed_ns = ELAPSED_NS(start, stop);
  fprintf(stdout, "Fluffed %.1f MBytes in %.2f ms (%.2f Gb/s)\n", buffsize*nloops/1.0e6, elapsed_ns / 1.0e6, (8 * (float)buffsize*nloops / elapsed_ns));

  test_buf(outbuf, buffsize);

  free(inbuf);
  free(outbuf);

  hipFree(d_inbuf);
  hipFree(d_outbuf);

  return EXIT_SUCCESS;
}

